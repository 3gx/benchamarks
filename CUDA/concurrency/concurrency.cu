#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <vector>
#include "rtc.h"
#include "cudamem.h"

#define __out

#define NTHREADS_MAX 1024

template<typename REAL>
__global__ void dev_compute(
    const int   nloop,
    __out REAL *in_a,
    const REAL *in_b,
    const REAL *in_c)
{
  const int tid = threadIdx.x;

  REAL       a = in_a[tid];
  const REAL b = in_b[tid];
  const REAL c = in_c[tid];

#pragma unroll 32
  for (int i = 0; i < nloop; i++)
    a += b*c;

  /* unlikely it will ever write result to RAM */
  if (a == REAL(123.123456))
    in_a[tid] = a;
}

template<typename T>
void run_test(const int nwarps, const int nblocks, const int n, const cuda_mem<T> &in, cuda_mem<T> &out)
{
  const int WARP_SIZE = 32;

  assert(nwarps > 0);
  assert(nblocks > 0);

  const dim3 grid(nblocks);
  const dim3 blocks(WARP_SIZE * nwarps);
  assert(blocks.x <= NTHREADS_MAX);

  const double t0 = rtc();

  dev_compute<T><<<grid, blocks>>>(n, out, in, in);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  const double t1 = rtc();
  const double dt = t1 - t0;

  fprintf(stderr, " nwarps= %d: done in %g sec perf= %g GFLOP/s\n", 
      nwarps, dt, 2.0*grid.x*n*blocks.x/dt/1e9 );
}


int main(int argc, char * argv[])
{
  const int nMloop = argc > 1 ? atoi(argv[1]) : 1;
  fprintf(stderr, " testing concurrency on %llu Melements\n", (unsigned long long)nMloop);

  const int nloop = nMloop * 1024;



  
  {
    fprintf(stderr, " --- fp32 ---  \n");
    cuda_mem<float> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);

    const int nblocks = 10240;

    for (int i = 1; i <= 32; i <<= 1)
    {
      run_test(i, nblocks, nloop, d_in, d_out);
    }
  }
  
  {
    fprintf(stderr, " --- fp64 ---  \n");
    cuda_mem<double> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);

    const int nblocks = 10240;

    for (int i = 1; i <= 32; i <<= 1)
    {
      run_test(i, nblocks, nloop, d_in, d_out);
    }
  }



  return 0;
}


