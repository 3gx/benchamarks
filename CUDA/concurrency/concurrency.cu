#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <vector>
#include "rtc.h"
#include "cudamem.h"

#define __out

#define NTHREADS_MAX 1024
#define WARP_SIZE 32

#define ILP 8

static __constant__ int warpSwitchOff[NTHREADS_MAX/WARP_SIZE];


template<typename REAL>
__global__ void dev_compute(
    const int nwarps,
    const int   nloop,
    __out REAL *in_a,
    const REAL *in_b,
    const REAL *in_c)
{
  const int tid = threadIdx.x;
//  if (tid/WARP_SIZE >= nwarps) return;
  if (!warpSwitchOff[tid / WARP_SIZE]) return;

  REAL       a[ILP] = {in_a[tid]};
  const REAL b      =  in_b[tid];
  const REAL c      =  in_c[tid];
  
#pragma unroll
  for (int i = 0; i < ILP; i++)
    a[i] += (float)i*a[i];

#pragma unroll 64
  for (int i = 0; i < nloop; i++)
  {
#pragma unroll
    for (int j = 0; j < ILP; j++)
      a[j] += b*c;
  }

  /* unlikely it will ever write result to RAM */
  REAL asum = a[0];
#pragma unroll
  for (int j = 1; j < ILP; j++)
    asum += a[j];
  if (asum == REAL(123.123456))
    in_a[tid] = asum;
}

  template<typename T>
void run_test(const int nwarps, const int warpStride, const int nblocks, const int n, const cuda_mem<T> &in, cuda_mem<T> &out)
{
  assert(nwarps > 0);
  assert(nblocks > 0);
  
  int warpMap_host[WARP_SIZE] = {0};
  int warpId = 0;
  int offset = 0;
  for (int i = 0; i < nwarps; i++)
  {
    warpMap_host[warpId + offset] = 1;
    warpId += warpStride;
    if (warpId + offset >= WARP_SIZE)
    {
      offset++;
      warpId -= WARP_SIZE;
    }
  }
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("warpSwitchOff"), warpMap_host, WARP_SIZE*sizeof(int)));

  const dim3 grid(nblocks);
  const dim3 blocks(NTHREADS_MAX);

  const double t0 = rtc();

  dev_compute<T><<<grid, blocks>>>(nwarps, n, out, in, in);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  const double t1 = rtc();
  const double dt = t1 - t0;

  fprintf(stderr, " nwarps= %d: done in %g sec perf= %g GFLOP/s\n", 
      nwarps, dt, ILP*2.0*grid.x*n*nwarps*WARP_SIZE/dt/1e9 );
}


int main(int argc, char * argv[])
{
  const int nMloop = argc > 1 ? atoi(argv[1]) : 1;
  fprintf(stderr, " testing concurrency on %llu Melements\n", (unsigned long long)nMloop);

  const int nloop = nMloop * 1024;

  const int warpStride = argc > 2 ? atoi(argv[2]) : 1;
  fprintf(stderr, " testing warp-stride of %d \n", warpStride);

  assert(warpStride > 0);

  int warpId = 0;
  int offset = 0;
  fprintf(stderr, " Warp scheduling order:  \n  ");
  for (int i = 0; i < WARP_SIZE; i++)
    fprintf(stderr, "%2d  ", i);
  fprintf(stderr, " \n  ");
  for (int i = 0; i < WARP_SIZE; i++)
  {
    fprintf(stderr, "%2d  ", warpId + offset);
    warpId += warpStride;
    if (warpId + offset >= WARP_SIZE)
    {
      offset++;
      warpId -= WARP_SIZE;
    }
  }
  fprintf(stderr, " \n");


  const int nblocks = 1;


  {
    fprintf(stderr, " --- fp32 ---  \n");
    cuda_mem<float> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);


    for (int i = 1; i <= NTHREADS_MAX/WARP_SIZE; i++)
    {
      run_test(i, warpStride, nblocks, nloop, d_in, d_out);
    }
  }

  {
    fprintf(stderr, " --- fp64 ---  \n");
    cuda_mem<double> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);

    for (int i = 1; i <= NTHREADS_MAX/WARP_SIZE; i++)
    {
      run_test(i, warpStride, nblocks, nloop, d_in, d_out);
    }
  }



  return 0;
}


