#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <vector>
#include "rtc.h"
#include "cudamem.h"

#define __out

#ifndef NTHREADS_MAX
#define NTHREADS_MAX 1024
#endif

#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

#define NWARPS (NTHREADS_MAX/WARP_SIZE)

#ifndef ILP
#define ILP 8
#endif

static __constant__ int warpSwitchOn[NWARPS];


template<typename REAL>
__global__ void dev_compute(
    const int nwarps,
    const int   nloop,
    __out REAL *in_a,
    const REAL *in_b,
    const REAL *in_c)
{
  const int tid = threadIdx.x;
  if (!warpSwitchOn[tid / WARP_SIZE]) return;

  REAL       a[ILP] = {in_a[tid]};
  const REAL b      =  in_b[tid];
  const REAL c      =  in_c[tid];
  
#pragma unroll
  for (int i = 0; i < ILP; i++)
    a[i] += (float)i*a[i];

#pragma unroll 64
  for (int i = 0; i < nloop; i++)
  {
#pragma unroll
    for (int j = 0; j < ILP; j++)
      a[j] += b*c;
  }

  /* unlikely it will ever write result to RAM */
  REAL asum = a[0];
#pragma unroll
  for (int j = 1; j < ILP; j++)
    asum += a[j];
  if (asum == REAL(123.123456))
    in_a[tid] = asum;
}

  template<typename T>
void run_test(const int nwarps, const std::vector<int> &warpList, const int nblocks, const int n, const cuda_mem<T> &in, cuda_mem<T> &out)
{
  assert(nwarps > 0);
  assert(nblocks > 0);
  
  int warpSwitchOn_host[NWARPS] = {0};
  for (int i = 0; i < nwarps; i++)
    warpSwitchOn_host[warpList[i]] = 1;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("warpSwitchOn"), warpSwitchOn_host, NWARPS*sizeof(int)));

  const dim3 grid(nblocks);
  const dim3 blocks(NTHREADS_MAX);

  const double t0 = rtc();

  dev_compute<T><<<grid, blocks>>>(nwarps, n, out, in, in);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  const double t1 = rtc();
  const double dt = t1 - t0;

  fprintf(stderr, " nwarps= %d: done in %g sec perf= %g GFLOP/s\n", 
      nwarps, dt, ILP*2.0*grid.x*n*nwarps*WARP_SIZE/dt/1e9 );
}


int main(int argc, char * argv[])
{
  assert( WARP_SIZE > 0);
  assert((WARP_SIZE & (WARP_SIZE-1)) == 0);
  const int nMloop = argc > 1 ? atoi(argv[1]) : 1;
  fprintf(stderr, " testing concurrency on %llu Melements\n", (unsigned long long)nMloop);

  const int nloop = nMloop * 1024;

  const int warpStride = argc > 2 ? atoi(argv[2]) : 1;
  fprintf(stderr, " testing warp-stride of %d \n", warpStride);

  assert(warpStride > 0);

  std::vector<int> warpList(NWARPS);

  fprintf(stderr, " Warp scheduling order:  \n  ");
  for (int i = 0; i < NWARPS; i++)
    fprintf(stderr, "%2d  ", i);
  fprintf(stderr, " \n  ");
  for (int i = 0; i < NWARPS; i++)
  {
    warpList[i] = i * warpStride;
    while (warpList[i] >= NWARPS)
      warpList[i] = (warpList[i] % NWARPS) + (warpList[i]/NWARPS);
    fprintf(stderr, "%2d  ", warpList[i]);
  }
  fprintf(stderr, " \n");


#if 0
  const int nblocks = 10240;
#else
  const int nblocks = 1;
#endif


  {
    fprintf(stderr, " --- fp32 ---  \n");
    cuda_mem<float> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);


    for (int i = 1; i <= NWARPS; i++)
    {
      run_test(i, warpList, nblocks, nloop, d_in, d_out);
    }
  }
  
  {
    fprintf(stderr, " --- double ---  \n");
    cuda_mem<double> d_in, d_out;

    d_in .realloc(NTHREADS_MAX);
    d_out.realloc(NTHREADS_MAX);

    for (int i = 1; i <= NWARPS; i++)
    {
      run_test(i, warpList, nblocks, nloop, d_in, d_out);
    }
  }

  return 0;
}


