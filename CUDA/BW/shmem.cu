#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <vector>
#include "rtc.h"
#include "cudamem.h"

#define __out

#ifdef FP64
typedef double real;
#else
typedef float real;
union float2x
{
  float2 flt;
  double dbl;
  __device__ float2x (const double x) : dbl(x) {}
  __device__ operator float2 () const { return flt; }
};
#endif


#define WARP_SIZE2 5

dim3 grid(const int nt, const int n)
{
  const int nb = (n-1)/nt + 1;
  dim3 grid(nb);
  if (grid.x > 65535)
  {
    grid.x = std::sqrt(nb);
    grid.y = (nb-1)/grid.x + 1;
  }
  return grid;
}

template<int M, int MODE>
__global__ void dev_compute(
    const int   n,
    const real *in,
    __out real *out)
{
  const int bid = blockIdx.y*gridDim.x + blockIdx.x;
  const int tid = bid * blockDim.x + threadIdx.x;

  real res = real(0.0);

  const int t0 = ((threadIdx.x >> WARP_SIZE2) << WARP_SIZE2) >> 1;
  volatile __shared__ real sxd[M];
  if (threadIdx.x < M)
    sxd[threadIdx.x] = in[t0+threadIdx.x];
  __syncthreads();

  if (MODE <= 3)
  {

    real xd[M];
#pragma unroll
    for (int i = 0; i < M; i++)
      xd[i] = sxd[i];

    switch(MODE)
    {
      case 0:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += xd[j]*xd[i];
        }
        break;

      case 1:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += sxd[j]*xd[i];
        }
        break;

      case 2:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += xd[j]*sxd[i];
        }
        break;

      case 3:
        const int laneId = threadIdx.x & 31;
        const real x = sxd[laneId];
        __syncthreads();

#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
          {
#ifdef SM30
            const real xi = __shfl(x, i, 1<<WARP_SIZE2);
            const real xj = __shfl(x, j, 1<<WARP_SIZE2);
#else
            const real xi = sxd[i];
            const real xj = sxd[j];
#endif
            res += xj*xi;
          }
        }
        break;
    }
  }
  else
  {
#if !defined FP64 && defined FP32OPT
    volatile double *sxd2 = (double*)sxd;

#pragma unroll
    for (int i = 0; i < M/2; i++)
    {
#pragma unroll
      for (int j = 0; j < M/2; j++)
      {
        const double x = sxd2[i];
        const double y = sxd2[j];
#if 1  /* compile with sm_30 to get performance */
        res += __int_as_float(__double2loint(x))*__int_as_float(__double2loint(y));
        res += __int_as_float(__double2loint(x))*__int_as_float(__double2hiint(y));
        res += __int_as_float(__double2hiint(x))*__int_as_float(__double2loint(y));
        res += __int_as_float(__double2hiint(x))*__int_as_float(__double2hiint(y));
#else
        const float2x xi(x);
        const float2x xj(y);
        res += xi.flt.x * xj.flt.x;
        res += xi.flt.x * xj.flt.y;
        res += xi.flt.y * xj.flt.x;
        res += xi.flt.y * xj.flt.y;
#endif
      }
    }

#else  /* naive, delivers only half of shmem bw here and quarter of performance... */

#pragma unroll
    for (int i = 0; i < M; i++)
    {
#pragma unroll
      for (int j = 0; j < M; j++)
      {
        res += sxd[i]*sxd[j];
      }
    }
#endif

  }


  /* unlikely it will ever write result to RAM */
  if (tid < n)
  {
#ifndef CKSUM
    if (res == real(123.123456))
#endif
      out[tid] = res;
  }
}

static inline void cksum(const cuda_mem<real> &d, host_mem<real> &h)
{
#ifdef CKSUM
  d.d2h(h);
  double cksum = 0.0;
  for (size_t i = 0; i < h.n; i++)
    cksum += h[i]*h[i];
  fprintf(stderr, " CKSUM= %16.15g  ", cksum);
#endif
}

int main(int argc, char * argv[])
{
  const size_t nMel = argc > 1 ? atoi(argv[1]) : 1;
  cuda_mem<real> d_in, d_out;
  host_mem<real> h_data;

  fprintf(stderr, " testing BW on %llu Melements\n", (unsigned long long)nMel);

  const size_t n = nMel * 1000000;

  h_data.realloc(n);
  d_in  .realloc(n);
  d_out .realloc(n);


  std::vector<real> h0(n);
  for (size_t i = 0; i < n; i++)
    h0[i] = drand48();

#ifdef FP64
  const int M = 16;
#else
  const int M = 32;
#endif
  {
    fprintf(stderr, " compute  REG - REG : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,0><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s  rgstr bw: %g GB/s\n", 
        n*M*M*2/dt/1e9,
        n*M*M*2.0*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " compute SHMEM- REG : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,1><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s  shmem bw: %g GB/s\n", 
        n*M*M*2/dt/1e9,
        n*M*M*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " compute  REG -SHMEM: ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,2><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s  shmem bw: %g GB/s\n", 
        n*M*M*2/dt/1e9,
        n*M*M*sizeof(real)/dt/1e9);
  }
#ifdef SM30
  {
    fprintf(stderr, " compute  SHFL-SHFL : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,3><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }
#endif
  {
    fprintf(stderr, " compute SHMEM-SHMEM: ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,255><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
#if !defined FP64 && defined FP32OPT
    fprintf(stderr, " %g GFLOP/s  shmem bw: %g GB/s\n", 
        n*M*M*2/dt/1e9,
        2.0*n*M*M/4.0*sizeof(double)/dt/1e9);
#else
    fprintf(stderr, " %g GFLOP/s  shmem bw: %g GB/s\n", 
        n*M*M*2/dt/1e9,
        2.0*n*M*M*sizeof(real)/dt/1e9);
#endif
  }


  return 0;
}


