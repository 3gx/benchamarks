#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <vector>
#include "rtc.h"
#include "cudamem.h"

#define __out

#ifdef FP64
typedef double real;
#else
typedef float real;
#endif

#define WARP_SIZE2 5

dim3 grid(const int nt, const int n)
{
  const int nb = (n-1)/nt + 1;
  dim3 grid(nb);
  if (grid.x > 65535)
  {
    grid.x = std::sqrt(nb);
    grid.y = (nb-1)/grid.x + 1;
  }
  return grid;
}

template<int M, int MODE>
__global__ void dev_compute(
    const int   n,
    const real *in,
    __out real *out)
{
  const int bid = blockIdx.y*gridDim.x + blockIdx.x;
  const int tid = bid * blockDim.x + threadIdx.x;

  real res = real(0.0);

  const int t0 = ((tid >> WARP_SIZE2) << WARP_SIZE2) >> 1;
  volatile __shared__ real sxd[M];
  if (tid < M)
    sxd[tid] = in[t0+tid];
  __syncthreads();

  if (MODE <= 3)
  {
    real xd[M];
#pragma unroll
    for (int i = 0; i < M; i++)
      xd[i] = sxd[i];

    switch(MODE)
    {
      case 0:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += xd[j]*xd[i];
        }
        break;

      case 1:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += sxd[j]*xd[i];
        }
        break;

      case 2:
#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
            res += xd[j]*sxd[i];
        }
        break;

      case 3:
        const int laneId = threadIdx.x & 31;
        const real x = sxd[laneId];
        __syncthreads();

#pragma unroll
        for (int i = 0; i < M; i++)
        {
#pragma unroll
          for (int j = 0; j < M; j++)
          {
#ifdef SM30
            const real xi = __shfl(x, i);
            const real xj = __shfl(x, j);
#else
            const real xi = sxd[i];
            const real xj = sxd[j];
#endif
            res += xj*xi;
          }
        }
        break;
    }
  }
  else
  {
#pragma unroll
    for (int i = 0; i < M; i++)
    {
#pragma unroll
      for (int j = 0; j < M; j++)
        res += sxd[j]*sxd[i];
    }
  }
  

  /* unlikely it will ever write result to RAM */
  if (tid < n)
  {
#ifndef CKSUM
    if (res == real(123.123456))
#endif
      out[tid] = res;
  }
}

static inline void cksum(const cuda_mem<real> &d, host_mem<real> &h)
{
#ifdef CKSUM
  d.d2h(h);
  double cksum = 0.0;
  for (size_t i = 0; i < h.n; i++)
    cksum += h[i]*h[i];
  fprintf(stderr, " CKSUM= %16.15g  ", cksum);
#endif
}

int main(int argc, char * argv[])
{
  const size_t nMel = argc > 1 ? atoi(argv[1]) : 1;
  cuda_mem<real> d_in, d_out;
  host_mem<real> h_data;

  fprintf(stderr, " testing BW on %llu Melements\n", (unsigned long long)nMel);

  const size_t n = nMel * 1000000;

  h_data.realloc(n);
  d_in  .realloc(n);
  d_out .realloc(n);


  std::vector<real> h0(n);
  for (int i = 0; i < n; i++)
    h0[i] = drand48();

#ifdef FP64
  const int M = 16;
#else
  const int M = 32;
#endif
  {
    fprintf(stderr, " compute  REG - REG : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,0><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }
  {
    fprintf(stderr, " compute SHMEM- REG : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,1><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }
  {
    fprintf(stderr, " compute  REG -SHMEM: ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,2><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }
#ifdef SM30
  {
    fprintf(stderr, " compute  SHFL-SHFL : ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,3><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }
#endif
  {
    fprintf(stderr, " compute SHMEM-SHMEM: ");
    for (size_t i = 0; i < n; i++)
      h_data[i] = h0[i];
    d_in.h2d(h_data);

    const double t0 = rtc();
    const int NTHREADS = 256;
    dev_compute<M,255><<<grid(NTHREADS,n), NTHREADS>>>(n, d_in, d_out);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt =  rtc() - t0;
    cksum(d_out, h_data);
    fprintf(stderr, " %g GFLOP/s\n", n*M*M*2/dt/1e9);
  }


  return 0;
}


