#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include "rtc.h"
#include "cudamem.h"

#define __out

typedef double real;

dim3 grid(const int nt, const int n)
{
  const int nb = (n-1)/nt + 1;
  dim3 grid(nb);
  if (grid.x > 65535)
  {
    grid.x = std::sqrt(nb);
    grid.y = (nb-1)/grid.x + 1;
  }
  return grid;
}

__global__ void dev_copy(
    const int   n,
    const real *in,
    __out real *out)
{
  const int bid = blockIdx.y*gridDim.x + blockIdx.x;
  const int tid = bid * blockDim.x + threadIdx.x;
  if (tid < n)
    out[tid] = in[tid];
}

int main(int argc, char * argv[])
{
  const size_t nMel = argc > 1 ? atoi(argv[1]) : 32;
  cuda_mem<real> d_data1, d_data2;
  host_mem<real> h_data1, h_data2;

  fprintf(stderr, " testing BW on %llu Melements\n", (unsigned long long)nMel);

  const size_t n = nMel * 1000000;

  h_data1.realloc(n);
  h_data2.realloc(n);

  real* h_data3 = (real*)malloc(sizeof(real)*n);

  /* first touch */
#pragma omp parallel for schedule(static)
  for (size_t i = 0; i < n; i++)
    h_data1[i] = h_data2[i] =  h_data3[i] = 0;

  d_data1.realloc(n);
  d_data2.realloc(n);


  int nthreads = 0;
#pragma omp parallel
#pragma omp critical
  nthreads++;

  fprintf(stderr, " Runing on %d OMP threads \n", nthreads);
  fprintf(stderr, " ------------------ \n");

  {
    fprintf(stderr, " host write BW pinned: ");
    const real   f = (real)argc;
    const double t0 = rtc();
#pragma omp parallel for schedule(static)
    for (size_t i = 0; i < n; i++)
      h_data1[i] = f;
    const double dt =  rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " host read  BW pinned: ");
    real f = real(0.0);
    const double t0 = rtc();
#pragma omp parallel for schedule(static)
    for (size_t i = 0; i < n; i++)
      f += h_data1[i];
    const double dt =  rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }

  fprintf(stderr, " ------------------ \n");
  
  {
    fprintf(stderr, " host write BW paged : ");
    const real   f = (real)argc;
    const double t0 = rtc();
#pragma omp parallel for schedule(static)
    for (size_t i = 0; i < n; i++)
      h_data3[i] = f;
    const double dt =  rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " host read  BW paged : ");
    real f = real(0.0);
    const double t0 = rtc();
#pragma omp parallel for schedule(static)
    for (size_t i = 0; i < n; i++)
      f += h_data3[i];
    const double dt =  rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  
  fprintf(stderr, " ------------------ \n");

  {
    fprintf(stderr, " host->device BW pinned: ");
    const double t0 = rtc();
    d_data1.h2d(h_data1);
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " host->device BW paged : ");
    const double t0 = rtc();
    d_data1.h2d(h_data3);
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  
  fprintf(stderr, " ------------------ \n");

  {
    fprintf(stderr, " device->host BW pinned: ");
    const double t0 = rtc();
    d_data1.d2h(h_data1);
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  {
    fprintf(stderr, " device->host BW paged : ");
    const double t0 = rtc();
    d_data1.d2h(h_data3);
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }
  
  fprintf(stderr, " ------------------ \n");

  {
    fprintf(stderr, " device<->device BW: ");
    const double t0 = rtc();
    dev_copy<<<grid(1024, n), 1024>>>(n, d_data1, d_data2);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", 2.0*n*sizeof(real)/dt/1e9);
  }

  {
    fprintf(stderr, " device->host zero copy BW: ");
    const double t0 = rtc();
    real *dh_data2;
    CUDA_SAFE_CALL(hipHostGetDevicePointer(&dh_data2, h_data2, 0));
    dev_copy<<<grid(1024, n), 1024>>>(n, d_data1, dh_data2);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    const double dt = rtc() - t0;
    fprintf(stderr, " %g GB/s\n", n*sizeof(real)/dt/1e9);
  }

  fprintf(stderr, " ------------------ \n");


  return 0;
}


